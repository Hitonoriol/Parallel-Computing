#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <iostream>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "matrix.hpp"

using namespace std;

Matrix<double> a, b, result;

/* CUDA error checking macro */
#define gpuErrAssert(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void multiply_matrices(
    const double* a,
    const double* b,
    double* result,
    const int result_width, const int height_b,
    const int width_a, const int width_b
) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k;
    double tmp = 0;
    for (k = 0; k < height_b; ++k) {
        tmp += a[i * width_a + k] * b[k * width_b + j];
    }
    result[i * result_width + j] = tmp;
}

void load_matrices(const string &path_a, const string &path_b)
{
    ifstream file_a(path_a), file_b(path_b);
    if (!file_a.is_open() || !file_b.is_open()) {
        cerr << "Error: Failed to open matrix files.\n";
        exit(1);
    }
    file_a >> a;
    file_b >> b;
    result.set_size(b.get_width(), a.get_height());

    cout << "Loaded matrices:\n";
    cout << " A (" << quoted(path_a) << "):\n" << a;
    cout << " B (" << quoted(path_b) << "):\n" << b;
    cout << "Allocated multiplication result matrix A * B = C ["
        << result.get_height() << "x" << result.get_width() << "]\n\n";
}

void check_cuda_devices()
{
    int nDevices;
    hipGetDeviceCount(&nDevices);

    if (nDevices == 0) {
        cerr << "Error: No CUDA-compatible devices found!\n";
        exit(1);
    }

    cout << "Available CUDA devices:\n";
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("(%i)  Device name: %s\n", i, prop.name);
        printf("     Type: %s\n", prop.integrated ? "integrated" : "discrete");
        printf("     Total memory: %d MB\n", prop.totalGlobalMem / 0x100000);
        printf("     Memory Bus Width: %d bits\n", prop.memoryBusWidth);
        printf("     Peak Memory Bandwidth: %f GB/s\n",
            2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    }
    cout << '\n';
}

int main(int argc, char **argv)
{
    check_cuda_devices();
    /* Load two matrices from files */
    if (argc >= 3)
        load_matrices({ argv[1] }, { argv[2] });
    else
        load_matrices("matrix_a.txt", "matrix_b.txt");

    auto size_a = sizeof(double) * a.size();
    auto size_b = sizeof(double) * b.size();
    auto result_size = sizeof(double) * result.size();

    /* Allocate device buffers */
    double* d_a, *d_b, *d_result;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_result, result_size);

    /* Transfer data from host to device memory */
    hipMemcpy(d_a, a.data(), size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size_b, hipMemcpyHostToDevice);

    /* Execute kernel */
    cout << "Starting CUDA kernel execution...\n";
    dim3 threads_per_block(result.get_width(), result.get_height());
    multiply_matrices<<<1, threads_per_block>>>(
        d_a, d_b, d_result,
        result.get_width(), b.get_height(),
        a.get_width(), b.get_width()
    );
    gpuErrAssert(hipPeekAtLastError());
    gpuErrAssert(hipDeviceSynchronize());
    cout << "Done executing!\n";

    // Transfer data back to host memory
    hipMemcpy(result.data(), d_result, result_size, hipMemcpyDeviceToHost);
    cout << "Multiplication result: \n" << result;

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
}